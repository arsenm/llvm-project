#include "hip/hip_runtime.h"
// Check that when we link a bitcode module into a file using
// -mlink-builtin-bitcode, we apply the same attributes to the functions in that
// bitcode module as we apply to functions we generate.
//
// In particular, we check that ftz and unsafe-math are propagated into the
// bitcode library as appropriate.

// Build the bitcode library.  This is not built in CUDA mode, otherwise it
// might have incompatible attributes.  This mirrors how libdevice is built.
// RUN: %clang_cc1 -x c++ -fconvergent-functions -emit-llvm-bc -DLIB \
// RUN:   %s -o %t.bc -triple nvptx-unknown-unknown

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc -o - \
// RUN:   -fcuda-is-device -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK --check-prefix=NOFTZ

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc \
// RUN:   -fdenormal-fp-math-f32=preserve-sign -o - \
// RUN:   -fcuda-is-device -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK --check-prefix=FTZ

// RUN: %clang_cc1 -x cuda %s -emit-llvm -mlink-builtin-bitcode %t.bc \
// RUN:   -fdenormal-fp-math-f32=preserve-sign -o - \
// RUN:   -fcuda-is-device -funsafe-math-optimizations -triple nvptx-unknown-unknown \
// RUN: | FileCheck %s --check-prefix=CHECK

// Wrap everything in extern "C" so we don't have to worry about name mangling
// in the IR.
extern "C" {
#ifdef LIB

// This function is defined in the library and only declared in the main
// compilation.
void lib_fn() {}

#else

#include "Inputs/hip/hip_runtime.h"
__device__ void lib_fn();
__global__ void kernel() { lib_fn(); }

#endif
}

// CHECK-NOT: convergent
// The kernel and lib function should have the same attributes.
// CHECK: define{{.*}} void @kernel() [[kattr:#[0-9]+]]
// CHECK: define internal void @lib_fn() [[fattr:#[0-9]+]]


// NOFTZ: attributes [[kattr]] = { mustprogress noinline norecurse nounwind optnone "no-trapping-math"="true" "stack-protector-buffer-size"="8" "target-features"="+ptx32,+sm_20" }
// NOFTZ: attributes [[fattr]] = { mustprogress noinline nounwind optnone "no-trapping-math"="true" "stack-protector-buffer-size"="8" "target-features"="+ptx32,+sm_20" }


// FTZ: attributes [[kattr]] = { mustprogress noinline norecurse nounwind optnone "denormal-fp-math-f32"="preserve-sign,preserve-sign" "no-trapping-math"="true" "stack-protector-buffer-size"="8" "target-features"="+ptx32,+sm_20" }
// FTZ: attributes [[fattr]] = { mustprogress noinline nounwind optnone "denormal-fp-math-f32"="preserve-sign,preserve-sign" "no-trapping-math"="true" "stack-protector-buffer-size"="8" "target-features"="+ptx32,+sm_20" }

