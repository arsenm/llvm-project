// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -emit-llvm \
// RUN:   -disable-llvm-passes -o - %s | FileCheck -allow-deprecated-dag-overlap -check-prefix DEVICE %s

// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:   -disable-llvm-passes -o - %s | \
// RUN:  FileCheck -allow-deprecated-dag-overlap -check-prefix HOST %s

#include "Inputs/hip/hip_runtime.h"

// DEVICE: Function Attrs: mustprogress noinline nounwind optnone{{$}}
// DEVICE-NEXT: define dso_local void @_Z3foov
__device__ void foo() {}

// HOST: Function Attrs: mustprogress noconvergent noinline nounwind optnone{{$}}
// HOST-NEXT: define dso_local void @_Z3barv

// DEVICE: Function Attrs: mustprogress noinline nounwind optnone{{$}}
// DEVICE-NEXT: define dso_local void @_Z3barv
__host__ __device__ void baz();
__host__ __device__ void bar() {
  // DEVICE: call void @_Z3bazv() [[CALL_ATTR:#[0-9]+]]
  baz();
  // DEVICE: call i32 asm "trap;", "=l"() [[ASM_ATTR:#[0-9]+]]
  int x;
  asm ("trap;" : "=l"(x));
  // DEVICE: call void asm sideeffect "trap;", ""() [[ASM_ATTR:#[0-9]+]]
  asm volatile ("trap;");
}

// DEVICE: declare void @_Z3bazv() [[BAZ_ATTR:#[0-9]+]]

// No noconvergent
// DEVICE: attributes [[BAZ_ATTR]] = { nounwind "{{.*}} }

// HOST: declare void @_Z3bazv() [[BAZ_ATTR:#[0-9]+]]
// HOST: attributes [[BAZ_ATTR]] = { {{.*}}noconvergent{{.*}} }
